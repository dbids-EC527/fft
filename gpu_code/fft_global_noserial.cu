#include "hip/hip_runtime.h"
/*
    nvcc -arch sm_35 fft_global.cu ./base_code/fft_2d.c -o fft_global
*/

#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <complex.h>
#include <time.h>
#include <string.h>
#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <hip/hip_complex.h>

//My own serial fft2d code
#include "fft_2d.h"

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
 
#define PI 3.1415926535897932384
typedef double complex cplx;

//Definitions which turn on and off test printing
//#define PRINT_GPU
#define PRINT_MATRIX

//Best performance occurs when the number of pixels is divisable by the number of threads
#define BLOCK_DIM 	    16
#define GRID_DIM	      128

#define CHECK_TOL          0.05
#define MINVAL             0.0
#define MAXVAL             10.0

//Function prototypes
void initializeArray(cplx *arr, int len, int seed);
double interval(struct timespec start, struct timespec stop);
void printArray(int rowLen, cplx* data);
void runIteration(int rowLen);

/*......CUDA Device Functions......*/
// FFT kernel per thread code
__global__ void FFT_Kernel (int rowLen, hipDoubleComplex* data) 
{
  int i, j, iters;
  
  //Interleave threads over a single block of the total array
  for (iters = 0; iters < 2000; iters++) 
  {
    for (i = blockIdx.x * blockDim.x + threadIdx.x; i < rowLen; i += blockDim.x*gridDim.x)
    {
      for (j = blockIdx.y * blockDim.y + threadIdx.y; j < rowLen; j += blockDim.y*gridDim.y)
      {
        //Reduce the current pixel
        if(i>0 && i<rowLen-1 && j>0 && j<rowLen-1)
        {
          data[i*rowLen+j] = hipCadd(data[i*rowLen+j], make_hipDoubleComplex(5, 0));
        }
      }
    }
  __syncthreads();
  }
}

/*......Host Code......*/
int main(int argc, char **argv)
{
  //Get the row length
  if (argc > 1) {
    int rowLen = atoi(argv[1]);
    printf("Running code for %dx%d matrix\n", rowLen, rowLen);
    runIteration(rowLen);
  }
  else 
  {
    printf("Running code for 1024x1024 matrix\n");
    runIteration(1024);
    printf("Running code for 2048x2048 matrix\n");
    runIteration(2048);
  }     
  
  return 0;
}

//Runs an iteration of GPU and CPU code for a given row length
void runIteration(int rowLen)
{
  // GPU Timing variables
  hipEvent_t start, stop, start_kernel, stop_kernel;
  float elapsed_gpu, elapsed_gpu_kernel;
  
  //Serial Timing variables:
  struct timespec time_start, time_stop;

  //Define local vars for checking correctness
  int i, j, errCount = 0, zeroCount = 0;
  float currDiff, maxDiff = 0;

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));

  // Define size of matricies
  size_t allocSize = rowLen * rowLen * sizeof(cplx);

  // Allocate matricies on host memory
  cplx *h_array                    = (cplx *) malloc(allocSize);
  cplx *h_serial_array             = (cplx *) malloc(allocSize);

  // Initialize the host arrays
  printf("\nInitializing the arrays ...");
  // Arrays are initialized with a known seed for reproducability
  initializeArray(h_array, rowLen*rowLen, 2453);
  initializeArray(h_serial_array, rowLen*rowLen, 2453);
#ifdef PRINT_MATRIX  
  printf("h_array:\n");
  printArray(rowLen, h_array);
  printf("h_serial_array\n");
  printArray(rowLen, h_serial_array);
#endif
  printf("\t... done\n\n");

  // Allocate arrays on GPU global memory
  //cplx *d_array;
  //CUDA_SAFE_CALL(hipMalloc((void **)&d_array, allocSize));
  hipDoubleComplex* d_array;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_array, rowLen*rowLen*sizeof(hipDoubleComplex)));
  for(int i = 0; i < rowLen*rowLen; i++)
  {
    d_array[i] = make_hipDoubleComplex(creal(h_array[i]), cimag(h_array[i]));
    CUDA_SAFE_CALL(hipPeekAtLastError());
  }
  
  // Start overall GPU timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Transfer the arrays to the GPU memory
  //CUDA_SAFE_CALL(hipMemcpy(d_array, h_array, allocSize, hipMemcpyHostToDevice));

  // Configure the kernel
  dim3 DimGrid(GRID_DIM, GRID_DIM, 1);    
  dim3 DimBlock(BLOCK_DIM, BLOCK_DIM, 1); 
  printf("Kernal code launching\n");

#ifdef PRINT_GPU
  cudaPrintfInit();
#endif

  // Start kernel timing
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  hipEventRecord(start_kernel, 0);

  // Compute the mmm for each thread
  FFT_Kernel<<<DimGrid, DimBlock>>>(rowLen, d_array);
  hipDeviceSynchronize();

  // End kernel timing
  hipEventRecord(stop_kernel, 0);
  hipEventSynchronize(stop_kernel);
  hipEventElapsedTime(&elapsed_gpu_kernel, start_kernel, stop_kernel);
  printf("\nGPU kernel time: %f (msec)\n", elapsed_gpu_kernel);   
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel); 

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  CUDA_SAFE_CALL(hipMemcpy(h_array, d_array, allocSize, hipMemcpyDeviceToHost));
  
#ifdef PRINT_GPU
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
#endif

  // End overall GPU timing
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  printf("GPU overall time: %f (msec)\n", elapsed_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Compute the results on the host
  printf("FFT_serial() start\n");
  clock_gettime(CLOCK_REALTIME, &time_start);
  fft_2d(h_serial_array, rowLen, rowLen*rowLen);
  clock_gettime(CLOCK_REALTIME, &time_stop);
  double time_spent = interval(time_start, time_stop);
  printf("FFT_serial() took %f seconds\n", time_spent);

  // Compare the results
#ifdef PRINT_MATRIX
  printf("GPU code:\n");
  printArray(rowLen, h_array);
  printf("serial code:\n");
  printArray(rowLen, h_serial_array);
#endif
  /*for(i = 0; i < rowLen; i++) {
    for(j = 0; j < rowLen; j++)
    {
        currDiff = abs(h_serial_array[i] - h_array[i]);
	      maxDiff = (maxDiff < currDiff) ? currDiff : maxDiff;
        if (currDiff > CHECK_TOL) {
            errCount++;
        }
        if (h_array[i] == 0) {
            zeroCount++;
        }
    }
  }
  
  if (errCount > 0) {
    float percentError = ((float)errCount / (float)(rowLen*rowLen)) * 100.0;
    printf("\n@ERROR: TEST FAILED: %d results did not match (%0.6f%%)\n", errCount, percentError);
  }
  else if (zeroCount > 0){
    printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
  }
  else {
    printf("\nTEST PASSED: All results matched\n");
  }
  printf("MAX_DIFFERENCE = %f between serial and GPU code\n\n", maxDiff);*/
  
  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(d_array));

  free(h_serial_array);
  free(h_array);

  CUDA_SAFE_CALL(hipDeviceReset());

  printf("Done with %dx%d matrix\n\n", rowLen, rowLen);
}

//Initiaizes the array to consistent random values
void initializeArray(cplx *arr, int len, int seed) {
  int i;
  float randNum;
  srand(seed);

  for (i = 0; i < len; i++) {
    randNum = ((float)rand()) / (float) RAND_MAX;
    arr[i] = (cplx)(MINVAL + (randNum * (MAXVAL - MINVAL)));
  }
}

//Calculates time interval for serial calculation
double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

//Print the array for testing
void printArray(int rowLen, cplx* data)
{
  long int i, j;
  for (i = 0; i < rowLen; i++)
  { 
    for (j = 0; j < rowLen; j++)
    { 
      printf("%.1f+j%.1f",creal(data[i*rowLen+j]), cimag(data[i*rowLen+j]));
    }
    printf("\n");
  }
}
