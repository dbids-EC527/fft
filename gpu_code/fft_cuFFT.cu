#include "hip/hip_runtime.h"
/*
    nvcc -arch compute_70 -code sm_70 fft_cuFFT.cu -o fft_cuFFT -lcufft 

    Helpful GPU code for reference:
    https://github.com/marianhlavac/FFT-cuda/blob/master/src/fft-cuda.cu
*/

#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <complex.h>
#include <time.h>
#include <string.h>
#include "./utilities/cuPrintf.cu"
#include "./utilities/cuPrintf.cuh"
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfftw.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
 
typedef double complex cplx;

//Definitions which turn on and off test printing
//#define PRINT_GPU
//#define PRINT_MATRIX

//Best performance occurs when the number of pixels is divisable by the number of threads
//Maximum Threads per Block is 1024, Maximum Shared Memory is 48KB
//cuComplexDouble is 16 bytes, therefore we can have 3072 elements in shared memory at once
#define MAX_SM_ELEM_NUM	  3072
#define BLOCK_DIM 	      16	 //Max of 32
#define GRID_DIM	        3072 //Max of 2147483647

#define CHECK_TOL          0.05
#define MINVAL             0.0
#define MAXVAL             10.0

//Function prototypes
void initializeArray(cplx *arr, int len, int seed);
double interval(struct timespec start, struct timespec stop);
void printArray(int rowLen, cplx* data);
void runIteration(int rowLen);
void show_buffer(cplx buf[], int rowLen, int n);
void transpose(cplx buf[], int rowLen);
void fft(cplx buf[], int n);
void fft_2d(cplx buf[], int rowLen, int n);

/*......Host Code......*/
int main(int argc, char **argv)
{
  //Get the row length
  if (argc > 1) 
  {
    int rowLen = atoi(argv[1]);
    printf("Running code for %dx%d matrix\n", rowLen, rowLen);
    runIteration(rowLen);
  }
  else 
  {
    for(int i = 2; i < 3072; i <<= 1)
    {
      printf("Running code for %dx%d matrix\n", i, i);
      runIteration(i);
    }
}     
  
  return 0;
}

//Runs an iteration of GPU and CPU code for a given row length
void runIteration(int rowLen)
{
  // GPU Timing variables
  hipEvent_t start, stop, start_kernel, stop_kernel;
  float elapsed_gpu, elapsed_gpu_kernel;
  
  //Serial Timing variables:
  struct timespec time_start, time_stop;

  //Define local vars for checking correctness
  int i, j, errCount = 0;
  double currDiff_real, currDiff_imag, maxDiff = 0;

  //Check that row can fit into SM
  if(rowLen > MAX_SM_ELEM_NUM)
  {
    fprintf(stderr, "The specified array will not work with shared memory\n");
    exit(EXIT_FAILURE);
  } 

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));

  // Define size of matricies
  size_t n = rowLen * rowLen;
  size_t allocSize = n * sizeof(cplx);
  
  // Allocate matricies on host memory
  cplx *h_array                    = (cplx *) malloc(allocSize);
  cplx *h_serial_array             = (cplx *) malloc(allocSize);

  // Initialize the host arrays
  printf("\nInitializing the arrays ...");
  // Arrays are initialized with a known seed for reproducability
  initializeArray(h_array, n, 2453);
  initializeArray(h_serial_array, n, 2453);
#ifdef PRINT_MATRIX  
  printf("h_array:\n");
  printArray(rowLen, h_array);
  printf("h_serial_array\n");
  printArray(rowLen, h_serial_array);
#endif
  printf("\t... done\n\n");

  //Copy double complex array to hipDoubleComplex array
  hipfftDoubleComplex* d = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex) * n);
  for(i = 0; i < n; i++)
  {
    double real_part = creal(h_array[i]);
    double imag_part = cimag(h_array[i]);
    d[i] = make_hipDoubleComplex(real_part, imag_part);
    CUDA_SAFE_CALL(hipPeekAtLastError());
  }

  // Allocate arrays on GPU global memory using cuFFT syntax
  hipfftHandle plan;
  hipfftDoubleComplex *d_array;
  //hipDoubleComplex* d_array;
  //hipDoubleComplex* d_array_out;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_array, n*sizeof(hipfftDoubleComplex)));
  //CUDA_SAFE_CALL(hipMalloc((void**)&d_array_out, n*sizeof(hipDoubleComplex)));
  
  // Start overall GPU timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Plan the cuFFT
  if(hipfftPlan2d(&plan, rowLen, rowLen, HIPFFT_Z2Z))
  {
    fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	  exit(EXIT_FAILURE);
  };

  //Transfer cuDoubleArray to device memory
  CUDA_SAFE_CALL(hipMemcpy(d_array, d, allocSize, hipMemcpyHostToDevice));

  // Configure the kernel
  dim3 DimGrid(GRID_DIM, 1, 1);      
  dim3 DimBlock(BLOCK_DIM, BLOCK_DIM, 1); 
  printf("Kernal code launching\n");

#ifdef PRINT_GPU
  cudaPrintfInit();
#endif

  // Start kernel timing
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  hipEventRecord(start_kernel, 0);

  // Compute the fft for each thread
  if (hipfftExecZ2Z(plan, d_array, d_array, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
    return;		
  }
  // int s = (int)log2((float)rowLen);

  // FFT_Kernel_Row<<<DimGrid, DimBlock>>>(rowLen, s, d_array_out, d_array);
  // hipDeviceSynchronize();

  // FFT_Kernel_Col<<<DimGrid, DimBlock>>>(rowLen, s, d_array, d_array_out);
  // hipDeviceSynchronize();

  // End kernel timing
  hipEventRecord(stop_kernel, 0);
  hipEventSynchronize(stop_kernel);
  hipEventElapsedTime(&elapsed_gpu_kernel, start_kernel, stop_kernel);
  printf("\nGPU kernel time: %f (msec)\n", elapsed_gpu_kernel);   
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel); 

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  CUDA_SAFE_CALL(hipMemcpy(d, d_array, allocSize, hipMemcpyDeviceToHost));
  
#ifdef PRINT_GPU
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
#endif

  // End overall GPU timing
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  printf("GPU overall time: %f (msec)\n", elapsed_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy hipDoubleComplex array to double complex array  
  for(i = 0; i < n; i++)
  {
    double real_part = hipCreal(d[i]);
    double imag_part = hipCimag(d[i]);
    h_array[i] = real_part + I*imag_part;
    CUDA_SAFE_CALL(hipPeekAtLastError());
  }
  
  // Compute the results on the host
  printf("FFT_serial() start\n");
  clock_gettime(CLOCK_REALTIME, &time_start);
  fft_2d(h_serial_array, rowLen, n);
  clock_gettime(CLOCK_REALTIME, &time_stop);
  double time_spent = interval(time_start, time_stop);
  printf("FFT_serial() took %f (msec)\n", time_spent*1000);
  
  // Compare the results
#ifdef PRINT_MATRIX
  printf("GPU code:\n");
  printArray(rowLen, h_array);
  printf("serial code:\n");
  printArray(rowLen, h_serial_array);
#endif
  for(i = 0; i < rowLen; i++) {
    for(j = 0; j < rowLen; j++)
    {
      currDiff_real = abs(creal(h_serial_array[i*rowLen+j]) - creal(h_array[i*rowLen+j]));
      currDiff_imag = abs(cimag(h_serial_array[i*rowLen+j]) - cimag(h_array[i*rowLen+j]));
      maxDiff = (maxDiff < currDiff_real) ? currDiff_real : maxDiff;
      maxDiff = (maxDiff < currDiff_imag) ? currDiff_imag : maxDiff;
      if (currDiff_real > CHECK_TOL || currDiff_imag > CHECK_TOL) {
        errCount++;	    
      }
    }
  }
  if (errCount > 0) {
    float percentError = ((float)errCount / (float)(n)) * 100.0;
    printf("\n@ERROR: TEST FAILED: %d results did not match (%0.6f%%)\n", errCount, percentError);
  }
  else {
    printf("\nTEST PASSED: All results matched\n");
  }
  printf("MAX_DIFFERENCE = %f between serial and GPU code\n\n", maxDiff);
  
  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(d_array));
  //CUDA_SAFE_CALL(hipFree(d_array_out));
  free(h_serial_array);
  free(h_array);
  hipfftDestroy(plan);

  CUDA_SAFE_CALL(hipDeviceReset());

  printf("Done with %dx%d matrix\n\n", rowLen, rowLen);
}

//Initiaizes the array to consistent random values
void initializeArray(cplx *arr, int len, int seed) {
  int i;
  float randNum;
  srand(seed);

  for (i = 0; i < len; i++) {
    randNum = ((float)rand()) / (float) RAND_MAX;
    arr[i] = (cplx)(MINVAL + (randNum * (MAXVAL - MINVAL)));
  }
}

//Calculates time interval for serial calculation
double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

//Print the array for testing
void printArray(int rowLen, cplx* data)
{
  long int i, j;
  for (i = 0; i < rowLen; i++)
  { 
    for (j = 0; j < rowLen; j++)
    { 
      if(cimag(data[i*rowLen+j]) < 0)
        printf("%.1f-j%.1f, ", creal(data[i*rowLen+j]), abs(cimag(data[i*rowLen+j])));
      else
      	printf("%.1f+j%.1f, ", creal(data[i*rowLen+j]), cimag(data[i*rowLen+j]));
    }
    printf("\n");
  }
}

/* Performs in place FFT on buf of size n*/
void fft(cplx buf[], int n) 
{
	//Rearrange the array such that it can be iterated upon in the correct order
	//This is called decimination-in-time or Cooley-Turkey algorithm to rearrange it first, then do nlogn iterations
	int i, j, len;
	for (i = 1, j = 0; i < n; i++) 
	{
		int bit = n >> 1;
		for (; j & bit; bit >>= 1)
				j ^= bit;
		j ^= bit;

		//swap(buf[i], buf[j]);
		cplx temp;
    if (i < j)
		{
			temp = buf[i];
			buf[i] = buf[j];
			buf[j] = temp;
		}
  }

	/*Compute the FFT for the array*/
	cplx w, u, v;
	// len goes 2, 4, ... n/2, n
	// len iterates over the array log2(n) times
  for (len = 2; len <= n; len <<= 1) 
	{
		double ang = 2 * M_PI / len;

		/* i goes from 0 to n with stride len
		j goes from 0 to len/2 in stride 1

		The sum of i+j is used to index into the buffer 
		and determine the correct indexes at which to perform the DFT.
		For example if n = 8:
		For the first iteration len = 2, i = 0,2,4,8, j = 0 so that i + j = 0,2,4,8.  
		For the second iteration len = 4, i = 0,4, j = 0,1  so that i + j = 0,1,4,5.  
		For the final iteration len = 8, i = 0, j = 0,1,2,3 so that i + j = 0,1,2,3.
		This allows us to DFT properly for each index based on the conceptual algorithm.

		For each iteration of there are n/2 iterations as shown above,
		*/
		for (i = 0; i < n; i += len) 
		{
			for (j = 0; j < (len / 2); j++) 
			{
				//Compute the DFT on the correct elements
				w = cexp(-I * ang * j);
        u = buf[i+j];
				v = buf[i+j+(len/2)] * w;
				buf[i+j] = u + v;
				buf[i+j+(len/2)] = u - v;
			}
		}
  }
}

/* Transpose the matrix */
void transpose(cplx buf[], int rowLen)
{
	int i, j;
	cplx temp;
	for (i = 0; i < rowLen; i++)
	{
		for (j = i+1; j < rowLen; j++)
		{
			temp = buf[i*rowLen + j];
			buf[i*rowLen + j] = buf[j*rowLen + i];
			buf[j*rowLen + i] = temp;
		}
	}
}

/* Orchestrates the row-column 2D FFT algorithm */
void fft_2d(cplx buf[], int rowLen, int n)
{
	// Do rows
	int i;
	for(i = 0; i < n; i += rowLen)
	{
		fft(buf+i, rowLen);
	}

	// Transpose the matrix
	transpose(buf, rowLen);

	// Do columns
	for(i = 0; i < n; i += rowLen)
	{
		fft(buf+i, rowLen);
	}

	// Transpose back
	transpose(buf, rowLen);
}
